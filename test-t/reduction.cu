#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define ROWS 3
#define COLUMNS 4

__global__ void reduction_cuda(float * d_out, const float * d_in)
{
  int tid = threadIdx.x+blockDim.x*blockIdx.x;

  if(tid<COLUMNS)
  {
    for(int j=0; j<ROWS; j++)
    {
      d_out[tid] = d_out[tid] + d_in[tid+COLUMNS*j];
    }
  }
}

int main()
{
    float array[ROWS][COLUMNS];
    for (int a=0; a<ROWS; a++)
    {
      for (int b=0; b<COLUMNS; b++)
      {
        array[a][b] = b+a*COLUMNS;
        printf("array[%d][%d] = %.2f; ", a, b, array[a][b]);
      }
    printf("\n");
    }
    printf("\n");
    float *d_in;
    hipMalloc((void**)&d_in,  COLUMNS*ROWS*sizeof(float));
    hipMemcpy(d_in, array, COLUMNS*ROWS*sizeof(float), hipMemcpyHostToDevice);
    float *d_out;
    hipMalloc((void**)&d_out, COLUMNS*sizeof(float));
    reduction_cuda << < ROWS, COLUMNS >> >(d_out, d_in);

    float *out;
    out = (float *)malloc(COLUMNS*sizeof(float));
    hipMemcpy(out, d_out,COLUMNS*sizeof(float), hipMemcpyDeviceToHost);
    printf("\n");
    for(int c=0; c<COLUMNS; c++)
    {
      printf("c[%d] = %.2f; ", c, out[c]);
    }
    printf("\n");
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
