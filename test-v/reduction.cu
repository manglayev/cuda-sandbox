#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define ROWS_A 2
#define COLUMNS_A_ROWS_B 3
//#define ROWS_B 3
#define COLUMNS_B 4
/*
Matrix multiplication
Simple parallel axb to bxc matrix multiplication
*/
__global__ void reduction_cuda(float * in_a, float * in_b, float * d_out)
{
  /*
  int tid_1 = threadIdx.x+blockDim.x*blockIdx.x;
  int tid_2 = threadIdx.x+blockDim.x*blockIdx.x;
  int tid_3 = threadIdx.x+blockDim.x*blockIdx.x;
  */
  int tid = threadIdx.x+blockDim.x*blockIdx.x;

  if (tid<ROWS_A)
  {
    if(tid<COLUMNS_B)
    {
      printf("tid = %d\n", tid);
      //printf("tid_1 = %d; tid_3 = %d\n", tid_1, tid_3);
    }
    //for(int i=0; i<COLUMNS_A_ROWS_B; i++)
    {
      //printf("integer division in GPU: %d / %d = %d;\n", tid_1, COLUMNS_A_ROWS_B, tid_1 / COLUMNS_A_ROWS_B);
      //d_out[tid_2] = d_out[tid_2] + in_a[(tid_1 / COLUMNS_A_ROWS_B)*COLUMNS_A_ROWS_B+i]*in_b[(i*COLUMNS_B)+tid_3];
    }
  }
}

int main()
{
    float array_1[ROWS_A][COLUMNS_A_ROWS_B];
    for (int a=0; a<ROWS_A; a++)
    {
      for (int b=0; b<COLUMNS_A_ROWS_B; b++)
      {
        array_1[a][b] = 1+a+b*COLUMNS_A_ROWS_B;
        printf("array_1[%d][%d] = %.2f; ", a, b, array_1[a][b]);
      }
    printf("\n");
    }
    printf("\n");
    float array_2[COLUMNS_A_ROWS_B][COLUMNS_B];
    for (int a=0; a<COLUMNS_A_ROWS_B; a++)
    {
      for (int b=0; b<COLUMNS_B; b++)
      {
        array_2[a][b] = 1+a+b*COLUMNS_B;
        printf("array_2[%d][%d] = %.2f; ", a, b, array_2[a][b]);
      }
    printf("\n");
    }
    printf("\n");

    float *in_a;
    hipMalloc((void**)&in_a,  ROWS_A*COLUMNS_A_ROWS_B*sizeof(float));
    hipMemcpy(in_a, array_1, ROWS_A*COLUMNS_A_ROWS_B*sizeof(float), hipMemcpyHostToDevice);

    float *in_b;
    hipMalloc((void**)&in_b,  COLUMNS_A_ROWS_B*COLUMNS_B*sizeof(float));
    hipMemcpy(in_b, array_2, COLUMNS_A_ROWS_B*COLUMNS_B*sizeof(float), hipMemcpyHostToDevice);

    float *d_out;
    hipMalloc((void**)&d_out, ROWS_A*COLUMNS_B*sizeof(float));
    reduction_cuda << < ROWS_A, COLUMNS_B >> >(in_a, in_b, d_out);

    float out[ROWS_A][COLUMNS_B];

    for (int a=0; a<ROWS_A; a++)
    {
      for (int b=0; b<COLUMNS_B; b++)
      {
        out[a][b] = 100;
      }
    }

    hipMemcpy(out, d_out, ROWS_A*COLUMNS_B*sizeof(float), hipMemcpyDeviceToHost);

    for (int a=0; a<ROWS_A; a++)
    {
      for (int b=0; b<COLUMNS_B; b++)
      {
    //    printf("out[%d][%d] = %.2f; ", a, b, out[a][b]);
      }
    printf("\n");
    }
    printf("\n");

    hipFree(in_a);
    hipFree(in_b);
    hipFree(d_out);

    return 0;
}
