
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 3
/*
  Matrix multiplication
  Simple parallel nxn to nxn matrix multiplication
*/
__global__ void matrixMultiplication(int * in_a, int * in_b, int * out_c)
{
  int index = threadIdx.x+blockDim.x*blockIdx.x;

  if(index < N*N)
  {
    for(int i=0; i<N; i++)
    {
      out_c[index] = out_c[index] + in_a[i+N*index/N] * in_b[index + N*i];
    }
  }
  /*
  int tid_1 = threadIdx.x+blockDim.x*blockIdx.x;
  int tid_2 = threadIdx.x+blockDim.x*blockIdx.x;
  int tid_3 = threadIdx.x+blockDim.x*blockIdx.x;
  if (tid<N)
  {
      //printf("tid = %d\n", tid);
      //printf("tid_1 = %d; tid_3 = %d\n", tid_1, tid_3);
      //for(int i=0; i<COLUMNS_A_ROWS_B; i++)
      //printf("integer division in GPU: %d / %d = %d;\n", tid_1, COLUMNS_A_ROWS_B, tid_1 / COLUMNS_A_ROWS_B);
      //d_out[tid_2] = d_out[tid_2] + in_a[(tid_1 / COLUMNS_A_ROWS_B)*COLUMNS_A_ROWS_B+i]*in_b[(i*COLUMNS_B)+tid_3];
  }
    */
}

int main()
{
    int array_1[N][N];
    for (int a=0; a<N; a++)
    {
      for (int b=0; b<N; b++)
      {
        array_1[a][b] = a+b;
        printf("array_1[%d][%d] = %d; ", a, b, array_1[a][b]);
      }
      printf("\n");
    }
    printf("\n");
    int array_2[N][N];
    for (int a=0; a<N; a++)
    {
      for (int b=0; b<N; b++)
      {
        array_2[a][b] = a+b;
        printf("array_2[%d][%d] = %d; ", a, b, array_2[a][b]);
      }
      printf("\n");
    }
    printf("\n");

    int *in_a;
    hipMalloc((void **)&in_a,  N*N*sizeof(int));
    hipMemcpy(in_a, array_1, N*N*sizeof(int), hipMemcpyHostToDevice);

    int *in_b;
    hipMalloc((void **)&in_b,  N*N*sizeof(int));
    hipMemcpy(in_b, array_2, N*N*sizeof(int), hipMemcpyHostToDevice);

    int *out_c;
    hipMalloc((void **)&out_c, N*N*sizeof(int));
    matrixMultiplication <<< N, N >>>(in_a, in_b, out_c);
    hipDeviceSynchronize();
    int out[N][N];
    for (int a=0; a<N; a++)
    {
      for (int b=0; b<N; b++)
      {
        out[a][b] = 1;
      }
    }
    hipMemcpy(out, out_c, N*N*sizeof(int), hipMemcpyDeviceToHost);
    printf("RESULT MATRIX:\n");
    for (int a=0; a<N; a++)
    {
      for (int b=0; b<N; b++)
      {
        printf("out[%d][%d] = %d; ", a, b, out[a][b]);
      }
      printf("\n");
    }
    printf("\n");

    hipFree(in_a);
    hipFree(in_b);
    hipFree(out_c);

    return 0;
}
