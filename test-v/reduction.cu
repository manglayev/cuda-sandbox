
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 4
/*
  Matrix multiplication
  Simple parallel nxn to nxn matrix multiplication
*/
__global__ void matrixMultiplication(int *in_a, int *in_b, int *out_c)
{
  int index = threadIdx.x+blockDim.x*blockIdx.x;

  if(index < N*N)
  {
    for(int i=0; i<N; i++)
    {
      out_c[index] = out_c[index] + in_a[i+(index/N)*N] * in_b[index-(index/N)*N+N*i];
    }
  }
}

int main()
{
    int *in_a, *in_b, *out_c;;
    hipMallocManaged(&in_a,  N*N*sizeof(int));
    hipMallocManaged(&in_b,  N*N*sizeof(int));
    hipMallocManaged(&out_c, N*N*sizeof(int));
    for (int a=0; a<N*N; a++)
    {
        in_a[a] = a;
        in_b[a] = a;
    }
    for (int a=0; a<N; a++)
    {
      for (int b=0; b<N; b++)
      {
        printf("in_a[%d][%d] = %d; ", a, b, in_a[a*N+b]);
      }
      printf("\n");
    }
    printf("\n");
    for (int a=0; a<N; a++)
    {
      for (int b=0; b<N; b++)
      {
        printf("in_b[%d][%d] = %d; ", a, b, in_b[a*N+b]);
      }
      printf("\n");
    }
    printf("\n");
    matrixMultiplication <<< N, N >>>(in_a, in_b, out_c);
    hipDeviceSynchronize();
    printf("RESULT MATRIX:\n");
    for (int a=0; a<N; a++)
    {
      for (int b=0; b<N; b++)
      {
        printf("out_c[%d][%d] = %d; ", a, b, out_c[b+a*N]);
      }
      printf("\n");
    }
    printf("\n");

    hipFree(in_a);
    hipFree(in_b);
    hipFree(out_c);

    return 0;
}
