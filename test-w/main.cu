#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define THREADS 5
#define BLOCKS 1

struct Column
{
  int a;
  int b[3];
};

__global__ void testFunction(float *dev_a)
{
  int thread = threadIdx.x;
  if(thread == 0)
  {
    dev_a[thread] = dev_a[thread]*dev_a[thread];
  }
}

int main()
{
  Column* columns = new Column[2];
  columns[0] = { 0, {1, 2, 3} };
  columns[1] = { 0, {4, 5, 6} };

  float a[THREADS] = { 1, 2, 3, 4, 5 };
  printf("BEFORE START 1\n");
  for(int i = 0; i<THREADS; i++)
    printf("a[%d] = %.2f; ", i, a[i]);
  printf("\nBEFORE END 2\n");
  float *dev_a;
  hipMalloc((void**)&dev_a, THREADS*sizeof(float));
  hipMemcpy(dev_a, a, THREADS*sizeof(float), hipMemcpyHostToDevice);
  testFunction<<<BLOCKS, THREADS>>>(dev_a);
  hipFree(dev_a);

  for(int c=0; c<3; c++)
    printf("columns[0].b[%d] = %d;\n", c, columns[0].b[c]);

    delete [] columns->Column::b;

  return 0;
}
